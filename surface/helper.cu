
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__inline__ __device__
float3 operator+(float3 a, float3 b)
{
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__inline__ __device__
float3 operator*(float s, float3 v)
{
    return make_float3(s*v.x, s*v.y, s*v.z);
}

__inline__ __device__
float3 lerp(float3 a, float3 b, float t)
{
    return (1.0-t) * a + t * b;
}

inline __device__
uint getGlobalIdx1d1d()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}
