#include "hip/hip_runtime.h"
#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
  #undef _GLIBCXX_ATOMIC_BUILTINS
  #undef _GLIBCXX_USE_INT128
#endif

#include "surface.h"
#include "mc_lut.h"
#include "helper.cu"
#include <iostream>


// Constant data
__constant__ dim3 VOL_DIM;


// Device Code
__inline__ __device__
uint getVertIdx(uint x, uint y, uint z)
{
    return x * VOL_DIM.y * VOL_DIM.z + y * VOL_DIM.z + z;
}

__inline__ __device__
uint getCellIdx(uint x, uint y, uint z)
{
    return x * (VOL_DIM.y -1) * (VOL_DIM.z -1) + y * (VOL_DIM.z -1) + z;
}

__inline __device__
uint3 getCellPos(uint idx, dim3 volDim)
{
    uint x = idx/ ((VOL_DIM.y -1) * (VOL_DIM.z -1));
    uint r = idx - (x * ((VOL_DIM.y -1) * (VOL_DIM.z -1)));
    uint y = r / (volDim.z -1);
    uint z = r - (y * (volDim.z -1));
    return make_uint3(x, y, z);
}


__global__
void triangulateCell(int* edges, int *lut,
                     float3 *volVerts, float *volSDF,
                     float3 *outTrianglePoints)
{
    uint cellIdx = getGlobalIdx1d1d();

    // Kill unused threads
    if (cellIdx > ((VOL_DIM.x -1) * (VOL_DIM.y -1) * (VOL_DIM.z -1)))
        return;
    
    dim3 pos = getCellPos(cellIdx, VOL_DIM);
    
    //printf("id: %d, x: %d, y: %d, z: %d\n", idx, pos.x, pos.y, pos.z);   
    
    // Get vertex indices of the cell;
    uint vertIdx[8];
    vertIdx[0] = getVertIdx(pos.x,   pos.y,   pos.z);
    vertIdx[1] = getVertIdx(pos.x+1, pos.y,   pos.z);
    vertIdx[2] = getVertIdx(pos.x+1, pos.y+1, pos.z);
    vertIdx[3] = getVertIdx(pos.x,   pos.y+1, pos.z);
    vertIdx[4] = getVertIdx(pos.x,   pos.y,   pos.z+1);
    vertIdx[5] = getVertIdx(pos.x+1, pos.y,   pos.z+1);
    vertIdx[6] = getVertIdx(pos.x+1, pos.y+1, pos.z+1);
    vertIdx[7] = getVertIdx(pos.x,   pos.y+1, pos.z+1);
    
    uint caseIdx = 0;
    for (uint i = 0; i < 8; i++) 
        caseIdx = caseIdx | (1 << i) * (volSDF[vertIdx[i]] <= 0.0);
    
    // Iterate edges                
    for (uint i = 0; i < 5; i++) { // triangle number
        for (uint j = 0; j < 3; j++) { // triangle point number
            uint offset = cellIdx*5*3 + i*3 + j;
            int edgeIdx = lut[caseIdx*5*3 + i*3 + j];
            
            if (edgeIdx == -1) {
                outTrianglePoints[offset]
                    = make_float3(offset, offset, offset);
                    continue;
            }

            int* edge = &edges[edgeIdx*2];
            
            //Global 1d vertex indices
            size_t vA = vertIdx[edge[0]];
            size_t vB = vertIdx[edge[1]];
            // Isolevel vales
            float isoA = volSDF[vA];
            float isoB = volSDF[vB];
            // Interpolate positions
            float alpha = isoA / (isoA - isoB);

            outTrianglePoints[offset] = lerp(volVerts[vA], volVerts[vB], alpha);
        }
    }
}

//
// Host code 
//

using namespace learnSPH;

std::vector<Eigen::Vector3d>
Surface::marchCubes(Eigen::Vector3i volDim,
                    std::vector<Eigen::Vector3f> &volVerts,
                    std::vector<float> &volSDF)
{
    assert(volVerts.size() == volSDF.size());

    const size_t VERT_COUNT = volVerts.size();
    const size_t CELL_COUNT = (volDim(0)-1)*(volDim(1)-1)*(volDim(2)-1);
    
    // Allocate device memory    
    float3 *d_volVerts;
    float *d_volSDF;
    int *d_lut;
    int *d_edges;
    hipMalloc((void **)&d_volVerts, VERT_COUNT * sizeof(float3));
    hipMalloc((void **)&d_volSDF, VERT_COUNT * sizeof(float));
    hipMalloc((void **)&d_lut, 256*5*3 * sizeof(int));
    hipMalloc((void **)&d_edges, 12*2 * sizeof(int));
    
    // Allocate unified memory
    float3* hd_outTriangles;
    hipMallocManaged(&hd_outTriangles,
                      3 * 5 * CELL_COUNT * sizeof(float3));
    
    // Copy memory to device
    hipMemcpy(d_volVerts, volVerts.front().data(),
               VERT_COUNT * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_volSDF, volSDF.data(),
               VERT_COUNT * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_lut, &MARCHING_CUBES_TABLE,
               256*5*3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, &CELL_EDGES,
               12*2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(VOL_DIM), volDim.data(), sizeof(dim3));

    // Run kernel
    uint blockSize = 256;
    uint gridSize = (CELL_COUNT + (blockSize -1)) /blockSize;
    //uint sharedMemSize = VERT_COUNT * sizeof(float3);

#ifndef NDEBUG
    std::cout << "BlockSize: " << blockSize << "\nGridSize: " <<  gridSize << std::endl;
#endif
    
    triangulateCell<<<gridSize, blockSize>>>(
        d_edges, d_lut,
        d_volVerts, d_volSDF,
        hd_outTriangles);

    hipDeviceSynchronize();
    
    // Reconstruct triangles
    std::vector<Eigen::Vector3d> triangles;
    for (uint c = 0; c < CELL_COUNT; c++) {
        for (uint i = 0; i < 5; i++) {
            for (uint j = 0; j < 3; j++) {
                uint offset = c*5*3 + i*3 + j;
                
                float3 point = hd_outTriangles[offset];
                                
                if (point.x == offset && point.y == offset && point.z == offset)
                    continue;

#ifndef NDEBUG
                std::cout << "N " << offset << ": "
                          << point.x << " " << point.y << " " << point.z << std::endl;
#endif                
                

                triangles.push_back(Eigen::Vector3d(point.x, point.y, point.z));
            }
        }
    }
    
#ifndef NDEBUG
    std::cout << "Number of Triangles: " << triangles.size() << std::endl;
#endif

    // Free device memory
    hipFree(&d_volVerts);
    hipFree(&d_volSDF);
    hipFree(&d_lut);
    hipFree(&d_edges);
    hipFree(&hd_outTriangles);

    return triangles;
}
